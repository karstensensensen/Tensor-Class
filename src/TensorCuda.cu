#include "hip/hip_runtime.h"
#include "TensorCudaBones.cuh"

namespace TSlib
{
	double_t round(double_t x, double_t place)
	{
		return double_t(int(x) * place) / place;
	}

	void CUDAInitialize(int device)
	{
		/*
		* this is primarily used to initialize the cuda api. This oftens takes some time to load so this function makes it possible to have more control over when this pause will happen.
		*/
		hipSetDevice(device);
		hipDeviceSynchronize();
		devcount = device;
		hipGetDeviceProperties(&props, devcount);

		#ifdef _TS_DEBUG
		CUDA_IS_INITIALIZED = true;
		#endif
	}
}